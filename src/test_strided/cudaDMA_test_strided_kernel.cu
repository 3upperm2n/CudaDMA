#include "hip/hip_runtime.h"
/*
 *  Copyright 2010 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

/* 
   Test kernel code for efficient DMAs
 */

#pragma once

#include <stdio.h>
#include "../../include/cudaDMA.h"

#define NUM_ITERS 1000
#define F4S_IN_SHMEM 256
#define NUM_ELEMENTS F4S_IN_SHMEM
#define NUM_DMA_THREADS 32
//#define DEBUG_PRINT 1

__global__ void
dma_4ld_strided( float4* g_idata, float4* g_odata, clock_t * timer_vals, const unsigned int el_cnt, const unsigned int el_size, const unsigned int el_stride ) 
{
//  cudaDMAStrided<128> dma0 (1, NUM_DMA_THREADS, NUM_ELEMENTS,
//	  NUM_ELEMENTS, el_size, el_cnt, el_stride, el_size);
//  cudaDMAStrided<128> dma1 (2, NUM_DMA_THREADS, NUM_ELEMENTS,
//	  NUM_ELEMENTS+NUM_DMA_THREADS, el_size, el_cnt, el_stride, el_size);
  cudaDMAStridedSmallElements dma0 (1, NUM_DMA_THREADS, NUM_ELEMENTS,
        NUM_ELEMENTS, el_size, el_cnt, el_stride, el_size);
  cudaDMAStridedSmallElements dma1 (2, NUM_DMA_THREADS, NUM_ELEMENTS,
        NUM_ELEMENTS+NUM_DMA_THREADS, el_size, el_cnt,
        el_stride, el_size);


  // shared memory
  __shared__  float4 sdata_i0[F4S_IN_SHMEM];
  __shared__  float4 sdata_i1[F4S_IN_SHMEM];
  // access thread id
  const unsigned int tid = threadIdx.x ;
//  if(tid == 0) printf("%u %u %u\n", el_size, el_cnt, el_stride);

  // Preamble
  float4 acc = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

  if (tid<F4S_IN_SHMEM) {
    sdata_i0[tid] = acc;
    sdata_i1[tid] = acc;
  }
  __syncthreads();


  if (tid<F4S_IN_SHMEM) {

    // Pre-amble:
    dma1.start_async_dma();
    dma0.start_async_dma();
    dma1.wait_for_dma_finish();
    float4 tmp1 = sdata_i1[tid];
    dma1.start_async_dma();
    acc.x += tmp1.x;
    acc.y += tmp1.y;
    acc.z += tmp1.z;
    acc.w += tmp1.w;

    for (unsigned int i = 0; i < NUM_ITERS-2; i+=2) {

      // Phase 1:
      dma0.wait_for_dma_finish();
      float4 tmp0 = sdata_i0[tid];
      dma0.start_async_dma();
      acc.x += tmp0.x;
      acc.y += tmp0.y;
      acc.z += tmp0.z;
      acc.w += tmp0.w;

      // Phase 2:
      dma1.wait_for_dma_finish();
      float4 tmp1 = sdata_i1[tid];
      dma1.start_async_dma();
      acc.x += tmp1.x;
      acc.y += tmp1.y;
      acc.z += tmp1.z;
      acc.w += tmp1.w;

    }

    // Postamble:
    dma0.wait_for_dma_finish();
    float4 tmp = sdata_i0[tid];
    acc.x += tmp.x;
    acc.y += tmp.y;
    acc.z += tmp.z;
    acc.w += tmp.w;
    g_odata[tid] = acc;

/*
    if(tid == 0) {
      printf("\n");
      for(unsigned int j = 0; j < NUM_ELEMENTS; j++)
        printf("%1.0f", sdata_i0[j].x);
      printf("\n");
      for(unsigned int j = 0; j < NUM_ELEMENTS; j++)
        printf("%1.0f", sdata_i1[j].x);
      printf("\n\n");
    }
*/

  } else if (dma0.owns_this_thread()) {
    for (unsigned int j = 0; j < NUM_ITERS; j+=2) {
      float4 * base_ptr = &g_idata[0];//[j*el_stride*el_cnt];
      dma0.execute_dma( base_ptr, &sdata_i0[0]);
    }
  } else if (dma1.owns_this_thread()) {
    for (unsigned int j = 1; j < NUM_ITERS; j+=2) {
      float4 * base_ptr = &g_idata[0];//[j*el_stride*el_cnt];
      dma1.execute_dma( base_ptr, &sdata_i1[0]);
    }
  }
}

