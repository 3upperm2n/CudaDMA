#include "hip/hip_runtime.h"
/*
 *  Copyright 2010 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

/* 
 * SAXPY example code using cudaDMA library.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

// includes, kernels
#include "saxpy_cudaDMA_kernel.cu"
#include "params.h"

#define MY_GPU_SM_COUNT 14
#define PRINT_ERRORS 1


#define CUDA_SAFE_CALL(x)					\
	{							\
		hipError_t err = (x);				\
		if (err != hipSuccess)				\
		{						\
			printf("Cuda error: %s\n", hipGetErrorString(err));	\
			exit(1);				\
		}						\
	}

void process_error( hipError_t error, char *string=0, bool verbose=false )
{
    if( hipSuccess != error )
    {
        if( string )
            printf( "%s: ", string );
        printf( "%s\n", hipGetErrorString( error ) );
        exit( -1 );
    }
        
    if( verbose && string )
        printf( "%s\n", string );
}


////////////////////////////////////////////////////////////////////////////////
void
computeGoldResults( float* y, float* x, float a, int num_elements) 
{
  for( unsigned int i = 0; i < num_elements; ++i) {
    y[i] = a * x[i] + y[i];
  }
}
////////////////////////////////////////////////////////////////////////////////

inline float frand() {
  return (float)rand()/(float)RAND_MAX;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{

  unsigned int num_elements = CTA_COUNT * NUM_ITERS * COMPUTE_THREADS_PER_CTA;
  unsigned int mem_size = sizeof(float) * num_elements;
  // Allocate host memory
  float* h_x = (float*) malloc( mem_size );
  float* h_y = (float*) malloc( mem_size );

  // Initalize the inputs
  srand(0);
  for( unsigned int i = 0; i < num_elements; ++i) {
    h_x[i] = frand();
    h_y[i] = frand();
  }
  float a = frand();

  // Allocate device memory
  float* d_x;
  float* d_y;
  CUDA_SAFE_CALL( hipMalloc( (void**) &d_x, mem_size));
  CUDA_SAFE_CALL( hipMalloc( (void**) &d_y, mem_size));
  // Copy host memory to device
  CUDA_SAFE_CALL( hipMemcpy( d_x, h_x, mem_size,
			     hipMemcpyHostToDevice) );
  CUDA_SAFE_CALL( hipMemcpy( d_y, h_y, mem_size,
			     hipMemcpyHostToDevice) );

  // Timer setup stuff
  hipError_t error = hipSuccess;
  float f_time_ms=0.f;
  hipEvent_t start, stop;
  error = hipEventCreate( &start );
  process_error( error, "create start event" );
  error = hipEventCreate( &stop );
  process_error( error, "create stop event" );
  
  clock_t * d_timer_vals;
  unsigned int timer_size = sizeof(clock_t) * 4 * num_elements;
  CUDA_SAFE_CALL( hipMalloc( (void**) &d_timer_vals, timer_size));

  // Execute the kernel
  if(BYTES_PER_DMA_THREAD * DMA_THREADS_PER_LD < DMA_SZ) printf("WARNING: Transfer is too large for templated value\n");
  if(ITERS_PER_COMPUTE_THREAD * COMPUTE_THREADS_PER_CTA < DMA_SZ_IN_FS) printf("WARNING: Compute threads will not process entire transfer\n");
  if(ITERS_PER_COMPUTE_THREAD * COMPUTE_THREADS_PER_CTA > DMA_SZ_IN_FS) printf("WARNING: Compute threads will overrun shared buffer if idx not checked\n");
  if(COMPUTE_THREADS_PER_CTA > DMA_SZ_IN_FS) printf("WARNING: Compute threads on first iter will overrun shared buffer if idx not checked\n");
  printf ("Launching kernel with:\n");
  printf ("   %d total CTAs per SM\n",(CTA_COUNT/MY_GPU_SM_COUNT));
  if ( (SAXPY_KERNEL==saxpy_cudaDMA) || (SAXPY_KERNEL==saxpy_cudaDMA_doublebuffer) ) {
    printf ("   %d total threads per CTA (%d compute, %d dma)\n",THREADS_PER_CTA,COMPUTE_THREADS_PER_CTA,DMA_THREADS_PER_CTA);
    printf ("   %d bytes per DMA thread\n",BYTES_PER_DMA_THREAD);
    printf ("   %d byte DMA transfer\n",DMA_SZ);
    printf ("   %d byte element size\n",EL_SZ);
    printf ("   %d elements\n",DMA_SZ/EL_SZ);
  } else {
    printf ("   %d total threads per CTA\n",THREADS_PER_CTA);
    printf ("   %d total warps per CTA\n",THREADS_PER_CTA/32);
  }
  hipEventRecord( start, 0 );
  SAXPY_KERNEL<<< CTA_COUNT, THREADS_PER_CTA >>>( d_y, d_x, a, d_timer_vals);
  error = hipDeviceSynchronize();
  hipEventRecord( stop, 0 );
  hipEventSynchronize(stop);
  process_error( error, "kernel" );
  error = hipEventElapsedTime( &f_time_ms, start, stop );
  process_error( error, "get event elapsed time" );

  // Stop Timer:
  printf( "Processing time: %f (ms)\n", f_time_ms);
  printf( "Bytes Processed: %d KB\n", static_cast<int>(static_cast<float>(mem_size) / 1024.0 ));

  // Allocate mem for the device results on host side
  float* h_results = (float*) malloc( mem_size );
  clock_t* h_timer_vals = (clock_t*) malloc (timer_size);
    
  // copy result from device to host
  CUDA_SAFE_CALL( hipMemcpy( h_results, d_y, mem_size, 
			     hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipMemcpy( h_timer_vals, d_timer_vals, timer_size, 
			     hipMemcpyDeviceToHost) );

#ifdef TIMERS_ON
  unsigned int warp0_time = (unsigned int) h_timer_vals[0];
  for (unsigned int i = 0; i < num_elements; i+=32) {
    //unsigned int diffclock = (unsigned int) h_timer_vals[num_threads+i] - (unsigned int) h_timer_vals[i];
    //printf("Warp %d iter cycles = %d\n",i,diffclock);
    printf("Warp %d clock1 = %d\n",i,(unsigned int) h_timer_vals[i] - warp0_time);
    printf("Warp %d clock2 = %d\n",i,(unsigned int) h_timer_vals[num_threads+i] - warp0_time);
    printf("Warp %d clock3 = %d\n",i,(unsigned int) h_timer_vals[2*num_threads+i] - warp0_time);
    printf("Warp %d clock4 = %d\n",i,(unsigned int) h_timer_vals[3*num_threads+i] - warp0_time);
  }
#endif

  // Compute and compare host reference results
  computeGoldResults( h_y, h_x, a, num_elements );
  bool res;
  res = true;
  for (unsigned int i = 0; i < num_elements; ++i) {
    // Not sure, but I think you can have rounding errors because GPUs support FMAs...
    if ((h_y[i]-h_results[i])>0.000001) {
      res = false;
#ifdef PRINT_ERRORS
      printf("ERROR: host y[%d] = %f\tdevice y[%d]=%f\tdifference = %f\n",i,h_y[i],i,h_results[i],h_results[i]-h_y[i]);
#endif
    } else {
      //printf(":)_:): host y[%d] = %f\tdevice y[%d]=%f\n",i,h_y[i],i,h_results[i]);
    }
  }
  if (res) {
    printf("PASSED\n");
  } else {
    printf("FAILED\n");
    printf("alpha = %f\n",a);
  }

  // Report bandwidths
  double d_log2_size = log2( static_cast<double> (mem_size/4) );
  printf( "Vector Dimension:  2^%.1f\n", d_log2_size);
  float f_bw = static_cast<float>(3*mem_size) / f_time_ms / static_cast<float>(1000000.0);
  printf( "Bandwidth: %f GB/s\n", f_bw);
  float f_gflops = static_cast<float>(2*(mem_size/4)) / f_time_ms / static_cast<float>(1000000.0);
  printf( "Performance: %f GFLOPS\n", f_gflops);
    
  // cleanup memory
  free( h_x );
  free( h_y );
  free( h_results );
  free( h_timer_vals );
  CUDA_SAFE_CALL(hipFree(d_x));
  CUDA_SAFE_CALL(hipFree(d_y));

  hipDeviceReset();
}

